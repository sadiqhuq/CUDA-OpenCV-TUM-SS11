#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    summer term 2011 / 19-26th September
*
* project: convolution
* file:    convolution_gpu.cu
*
* 
\********* PLEASE ENTER YOUR CORRECT STUDENT NAME AND ID BELOW **************/
const char* gpu_studentName = "Sadiq Huq";
const int   gpu_studentID   = 3273623;
/****************************************************************************\
*
* In this file the following methods have to be edited or completed:
*
* gpu_convolutionGrayImage_gm_d
* gpu_convolutionGrayImage_gm_cm_d
* gpu_convolutionGrayImage_sm_d
* gpu_convolutionGrayImage_sm_cm_d
* gpu_convolutionGrayImage_dsm_cm_d 
* gpu_convolutionInterleavedRGB_dsm_cm_d - Mode 5 interleaved 
* gpu_convolutionInterleavedRGB_tex_cm_d -  Model 6 interleaved
*
\****************************************************************************/


#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <iostream>


#include "convolution_gpu.cuh"


#define BW 16                 // block width
#define BH 16                 // block height
#define MAXKERNELSIZE 3000    // actual size, i.e. NOT the byte size





// constant memory block on device
__constant__ float constKernel[MAXKERNELSIZE];


// texture memory and descriptor
hipChannelFormatDesc tex_Image_desc = hipCreateChannelDesc<float>();
texture<float, 2, hipReadModeElementType> tex_Image;

hipChannelFormatDesc tex_Image_descF4 = hipCreateChannelDesc<float4>();
texture<float4, 2, hipReadModeElementType> tex_ImageF4;


__host__ const char* gpu_getStudentName() { return gpu_studentName; };
__host__ int         gpu_getStudentID()   { return gpu_studentID; };
__host__ bool        gpu_checkStudentNameAndID() { return strcmp(gpu_studentName, "John Doe") != 0 && gpu_studentID != 1234567; };




//----------------------------------------------------------------------------
// Gray Image Functions
//----------------------------------------------------------------------------


// mode 1: using global memory only
__global__ void gpu_convolutionGrayImage_gm_d(const float *inputImage, const float *kernel, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch, size_t kPitch)
{

  // ### implement me ### 
	int temp_x,temp_y;
	float sum;
	const int kWidth  = (kRadiusX<<1) + 1;
	const int kHeight = (kRadiusY<<1) + 1;
	
	const uint thposX = threadIdx.x + blockIdx.x * blockDim.x;
	const uint thposY = threadIdx.y + blockIdx.y * blockDim.y;

//	
	if (thposX >=iWidth || thposY >=iHeight)
		return;
	sum=0;
	for (int y=0;y<kHeight;y++)
	{
		for (int x=0;x<kWidth;x++)
		{
			temp_x= thposX + x - kRadiusX;
			temp_y= thposY + y - kRadiusY;
			
			if(temp_x < 0)
				temp_x=0;
			if(temp_x >= iWidth)
				temp_x=iWidth-1;
			if(temp_y < 0)
				temp_y=0;
			if(temp_y >= iHeight)
				temp_y=iHeight-1;

			sum = sum+kernel[y*kPitch+x]*inputImage[temp_x+temp_y*iPitch];
		}
	}
	outputImage[thposY*iPitch+thposX] = sum;	

}

// mode 2: using global memory and constant memory
__global__ void gpu_convolutionGrayImage_gm_cm_d(const float *inputImage, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch)
{

	int temp_x,temp_y;
	float sum;

	const int kWidth  = (kRadiusX<<1) + 1;
	const int kHeight = (kRadiusY<<1) + 1;

	const uint thposX = threadIdx.x + blockIdx.x * blockDim.x;
	const uint thposY = threadIdx.y + blockIdx.y * blockDim.y;


	if (thposX >=iWidth || thposY >=iHeight)
		return;
	sum=0;
	for (int y=0;y<kHeight;y++)
	{
		for (int x=0;x<kWidth;x++)
		{
			temp_x = thposX + x - kRadiusX;
			temp_y = thposY + y - kRadiusY;
			
			if(temp_x < 0)
				temp_x=0;
			if(temp_x >= iWidth)
				temp_x=iWidth-1;
			if(temp_y < 0)
				temp_y=0;
			if(temp_y >= iHeight)
				temp_y=iHeight-1;

			sum = sum+constKernel[y*kWidth+x]*inputImage[temp_x+temp_y*iPitch];
		}
	}
	outputImage[thposY*iPitch+thposX] = sum;		
}


// mode 3: using shared memory for image and global memory for kernel access
__global__ void gpu_convolutionGrayImage_sm_d(const float *inputImage, const float *kernel, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch, size_t kPitch)
{
  // make use of the constant MAXKERNELSIZE in order to define the shared memory size
  
	__shared__ float sdata[MAXKERNELSIZE];

	  const uint thposX = threadIdx.x + blockIdx.x * blockDim.x;
	  const uint thposY = threadIdx.y + blockIdx.y * blockDim.y;
	  
	  outputImage[thposX + thposY * iPitch] = 0;

	  const uint width = (kRadiusX << 1) + BW; 	
	  const uint height = (kRadiusY << 1) + BH;

	  const uint blockPixels = BW * BH;   		
	  const uint totalPixels = width * height;	

	  const uint startX = blockIdx.x * blockDim.x - kRadiusX;
	  const uint startY = blockIdx.y * blockDim.y - kRadiusY;

	  const uint threadPos = threadIdx.x + threadIdx.y * blockDim.x;

	  uint offset = 0;

	  while(offset < totalPixels)
	  {		    
		  int temp_x,temp_y;
		  temp_x=startX + ((threadPos + offset) % width);
		  temp_y=startY + ((threadPos + offset) / width);
		  if(temp_x < 0)
			  temp_x=0;
		  if(temp_x >= iWidth)
			  temp_x=iWidth-1;
		  if(temp_y < 0)
			  temp_y=0;
		  if(temp_y >= iHeight)
			  temp_y=iHeight-1;

		  if(offset + threadPos < totalPixels)  
			  sdata[offset + threadPos] = inputImage[temp_x + temp_y * iPitch];
		  offset += blockPixels; // stride
	  }
	  
	  __syncthreads();
	  
//	  outputImage[thposX + thposY * iPitch]=sdata[offset + blockPos];
	  
	    const int kWidth  = (kRadiusX<<1) + 1;
	  	const int kHeight = (kRadiusY<<1) + 1;
	  	
	  if(thposX < iWidth && thposY < iHeight)
	  {	    
	    float sum = 0.0f;
	          
	    const int blockWidth = blockDim.x + (kRadiusX<<1);
	    const int blockHeight = blockDim.y + (kRadiusY<<1);

	    for (int y = 0; y < kHeight; ++y)
	    	for (int x = 0; x < kWidth; ++x) {
	    		const int sharedY = threadIdx.y + y;
	    		const int sharedX = threadIdx.x + x;
	    		sum += sdata[sharedY * blockWidth + sharedX] * kernel[y * kPitch + x];
	    	}
	    outputImage[thposX + thposY * iPitch] = sum; 
    
	  }
}


// mode 4: using shared memory for image and constant memory for kernel access
__global__ void gpu_convolutionGrayImage_sm_cm_d(const float *inputImage, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch)
{
  // make use of the constant MAXKERNELSIZE in order to define the shared memory size
	  
			__shared__ float sdata[MAXKERNELSIZE];

		  // Global position of thread in image coordinates 
		  const uint thposX = threadIdx.x + blockIdx.x * blockDim.x;
		  const uint thposY = threadIdx.y + blockIdx.y * blockDim.y;
		  
		  outputImage[thposX + thposY * iPitch] = 0;  // Debug black

		  // apron => extra boundary space. width = block + apron region
		  const uint width = (kRadiusX << 1) + BW; 	
		  const uint height = (kRadiusY << 1) + BH;

		  // 16x16 number of pixels in block   BW = blockDim.x 
		  const uint blockPixels = BW * BH;   		
		  // number of pixels in (256 + apron pixels)
		  const uint totalPixels = width * height;	

		  // (block+apron)-region in image pixel coordinates  
		  // first pixel of each block, same for all threads in a particular block
		  const uint startX = blockIdx.x * blockDim.x - kRadiusX;
		  const uint startY = blockIdx.y * blockDim.y - kRadiusY;
		      
		  // y*rowsize + x => linear pos of the pixel thread in block
		  const uint threadPosInBlock = threadIdx.x + threadIdx.y * blockDim.x;
		 
		  uint offset = 0;

		  while(offset < totalPixels)
		  {		    
			  int temp_x,temp_y;
				temp_x=startX + ((threadPosInBlock + offset) % width);
				temp_y=startY + ((threadPosInBlock + offset) / width);
				if(temp_x < 0)
					temp_x=0;
				if(temp_x >= iWidth)
					temp_x=iWidth-1;
				if(temp_y < 0)
					temp_y=0;
				if(temp_y >= iHeight)
					temp_y=iHeight-1;
					
			    if(offset + threadPosInBlock < totalPixels)  
					      sdata[offset + threadPosInBlock] = inputImage[temp_x + temp_y * iPitch];
		    offset += blockPixels; 
		    
		  }
		  
		  __syncthreads();
		  
		    // convolution -  image on shared memory 
		  
		    const int kWidth  = (kRadiusX<<1) + 1;
		  	const int kHeight = (kRadiusY<<1) + 1;
		  	
		  if(thposX < iWidth && thposY < iHeight)
		  {
		    float sum = 0.0f;
		    for (int y = 0; y < kHeight; ++y)
		    	for (int x = 0; x < kWidth; ++x) {
		    		const int sharedY = threadIdx.y + y;
		    		const int sharedX = threadIdx.x + x;
		    		sum += sdata[sharedY * width + sharedX] * constKernel[y * kWidth + x];
		    	}

		    outputImage[thposX + thposY * iPitch] = sum; 	
		  }

} 



// mode 5: using dynamically allocated shared memory for image and constant memory for kernel access
__global__ void gpu_convolutionGrayImage_dsm_cm_d(const float *inputImage, float *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitch)
{

  // ### implement me ### 
	extern __shared__ float sdata[];

	  // Global position of thread in image coordinates 
	  const uint thposX = threadIdx.x + blockIdx.x * blockDim.x;
	  const uint thposY = threadIdx.y + blockIdx.y * blockDim.y;
	  
//	  outputImage[thposX + thposY * iPitch] = 0;  // Debug black

	  // apron => extra boundary space. width = block + apron region
	  const uint width = (kRadiusX << 1) + BW; 	
	  const uint height = (kRadiusY << 1) + BH;

	  // 16x16 number of pixels in block   BW = blockDim.x 
	  const uint blockPixels = BW * BH;   		
	  // number of pixels in (256 + apron pixels)
	  const uint totalPixels = width * height;	

	  // (block+apron)-region in image pixel coordinates  
	  // first pixel of each block, same for all threads in a particular block
	  const uint startX = blockIdx.x * blockDim.x - kRadiusX;
	  const uint startY = blockIdx.y * blockDim.y - kRadiusY;
	      
	  // y*rowsize + x => linear pos of the pixel thread in block
	  const uint threadPosInBlock = threadIdx.x + threadIdx.y * blockDim.x;
	 
	  uint offset = 0;

	  while(offset < totalPixels)
	  {		    
		  int temp_x,temp_y;
			temp_x=startX + ((threadPosInBlock + offset) % width);
			temp_y=startY + ((threadPosInBlock + offset) / width);
			if(temp_x < 0)
				temp_x=0;
			if(temp_x >= iWidth)
				temp_x=iWidth-1;
			if(temp_y < 0)
				temp_y=0;
			if(temp_y >= iHeight)
				temp_y=iHeight-1;
				
		    if(offset + threadPosInBlock < totalPixels)  
				      sdata[offset + threadPosInBlock] = inputImage[temp_x + temp_y * iPitch];
	    offset += blockPixels; 
	    
	  }
	  
	  __syncthreads();
	  
	    // convolution -  image on shared memory 
	  
	    const int kWidth  = (kRadiusX<<1) + 1;
	  	const int kHeight = (kRadiusY<<1) + 1;
	  	
	  if(thposX < iWidth && thposY < iHeight)
	  {
	    float sum = 0.0f;
	    for (int y = 0; y < kHeight; ++y)
	    	for (int x = 0; x < kWidth; ++x) {
	    		const int sharedY = threadIdx.y + y;
	    		const int sharedX = threadIdx.x + x;
	    		sum += sdata[sharedY * width + sharedX] * constKernel[y * kWidth + x];
	    	}

	    outputImage[thposX + thposY * iPitch] = sum; 	
	  }

} 


// mode 6: using texture memory for image and constant memory for kernel access
__global__ void gpu_convolutionGrayImage_tex_cm_d(const float *inputImage, float *outputImage,
    int iWidth, int iHeight, int kRadiusX, int kRadiusY,
    size_t iPitch)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= iWidth || y >= iHeight) return;
  
  const float xx = (float)x;
  const float yy = (float)y;
  const int kWidth = (kRadiusX<<1) + 1;

  int xk, yk;
  float value = 0.0f;

  for (xk=-kRadiusX; xk <= kRadiusX; xk++) {
    for (yk=-kRadiusY; yk <= kRadiusY; yk++) {
      value += tex2D(tex_Image,xx+xk,yy+yk) * constKernel[(yk+kRadiusY)*kWidth + xk+kRadiusX];
    }
  }

  outputImage[y*iPitch + x] = value;
}





void gpu_convolutionGrayImage(const float *inputImage, const float *kernel, float *outputImage, 
                              int iWidth, int iHeight, int kRadiusX, int kRadiusY, int mode)
{
  size_t iPitchBytes, kPitchBytes;
  size_t iPitch, kPitch;
  float *d_inputImage;
  float *d_kernel;
  float *d_outputImage;

  const int kWidth  = (kRadiusX<<1) + 1;
  const int kHeight = (kRadiusY<<1) + 1;

  assert(kWidth*kHeight <= MAXKERNELSIZE);
  
  // allocate device memory
  cutilSafeCall( hipMallocPitch( (void**)&d_inputImage, &iPitchBytes, iWidth*sizeof(float), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&d_outputImage, &iPitchBytes, iWidth*sizeof(float), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&d_kernel, &kPitchBytes, kWidth*sizeof(float), kHeight ) );  
  iPitch = iPitchBytes/sizeof(float);
  kPitch = kPitchBytes/sizeof(float);
  //std::cout << "iPitchBytes=" << iPitchBytes << " iPitch=" << iPitch << " kPitchBytes=" << kPitchBytes << " kPitch=" << kPitch << std::endl;
  
  cutilSafeCall( hipMemcpy2D(d_inputImage, iPitchBytes, inputImage, iWidth*sizeof(float), iWidth*sizeof(float), iHeight, hipMemcpyHostToDevice) );
  cutilSafeCall( hipMemcpy2D(d_kernel, kPitchBytes, kernel, kWidth*sizeof(float), kWidth*sizeof(float), kHeight, hipMemcpyHostToDevice) );


  gpu_bindConstantMemory(kernel, kWidth*kHeight);
  gpu_bindTextureMemory(d_inputImage, iWidth, iHeight, iPitchBytes);

  dim3 blockSize(BW,BH);  
  dim3 gridSize( ((iWidth%BW) ? (iWidth/BW+1) : (iWidth/BW)), ((iHeight%BH) ? (iHeight/BH+1) : (iHeight/BH)) );

  
  // invoke the kernel of your choice here
  const int smSize =  (blockSize.x+(kRadiusX<<1)) * (blockSize.y+(kRadiusY<<1)) * sizeof(float);  

  switch(mode) {
    case 1:
      gpu_convolutionGrayImage_gm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
      break;
    case 2:
      gpu_convolutionGrayImage_gm_cm_d<<<gridSize,blockSize,smSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
      break;
    case 3:
      gpu_convolutionGrayImage_sm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
      break;
    case 4:
      gpu_convolutionGrayImage_sm_cm_d<<<gridSize,blockSize,smSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
      break;
    case 5:
      gpu_convolutionGrayImage_dsm_cm_d<<<gridSize,blockSize,smSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
      break;
    case 6:
      gpu_convolutionGrayImage_tex_cm_d<<<gridSize,blockSize,smSize>>>(d_inputImage, d_outputImage,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
      break;
    default:
      std::cout << "gpu_convolutionGrayImage() Warning: mode " << mode << " is not supported." << std::endl;
  }

  cutilSafeCall( hipDeviceSynchronize() );
  cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float), d_outputImage, iPitchBytes, iWidth*sizeof(float), iHeight, hipMemcpyDeviceToHost) );
  

  // free memory
  gpu_unbindTextureMemory();
  cutilSafeCall( hipFree(d_inputImage) );
  cutilSafeCall( hipFree(d_outputImage) );
  cutilSafeCall( hipFree(d_kernel) );
}




//----------------------------------------------------------------------------
// RGB Image Functions (for separated color channels)
//----------------------------------------------------------------------------



void gpu_convolutionRGB(const float *inputImage, const float *kernel, float *outputImage, 
                        int iWidth, int iHeight, int kRadiusX, int kRadiusY, int mode)
{
  const int imgSize = iWidth*iHeight;
  gpu_convolutionGrayImage(inputImage, kernel, outputImage, iWidth, iHeight, kRadiusX, kRadiusY, mode);
  gpu_convolutionGrayImage(inputImage+imgSize, kernel, outputImage+imgSize, iWidth, iHeight, kRadiusX, kRadiusY, mode);
  gpu_convolutionGrayImage(inputImage+(imgSize<<1), kernel, outputImage+(imgSize<<1), iWidth, iHeight, kRadiusX, kRadiusY, mode);
}




//----------------------------------------------------------------------------
// RGB Image Functions (for interleaved color channels)
//----------------------------------------------------------------------------



// mode 5 (interleaved): using shared memory for image and constant memory for kernel access
__global__ void gpu_convolutionInterleavedRGB_dsm_cm_d(const float3 *inputImage, float3 *outputImage,
                                              int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                              size_t iPitchBytes)
{
	extern __shared__ float3 sdata3[];

	// Global position of thread in image coordinates 
	const uint thposX = threadIdx.x + blockIdx.x * blockDim.x;
	const uint thposY = threadIdx.y + blockIdx.y * blockDim.y;

	//	  outputImage[thposX + thposY * iPitch] = 0;  // Debug black

	// apron => extra boundary space. width = block + apron region
	const uint width = (kRadiusX << 1) + BW; 	
	const uint height = (kRadiusY << 1) + BH;

	const uint blockPixels = BW * BH;   		// 16x16 number of pixels in block   BW = blockDim.x 
	const uint totalPixels = width * height;	// number of pixels in (256 + apron pixels)

	// (block+apron)-region in image pixel coordinates  
	// first pixel of each block, same for all threads in a particular block
	const uint startX = blockIdx.x * blockDim.x - kRadiusX;
	const uint startY = blockIdx.y * blockDim.y - kRadiusY;

	// x + y*rowsize  => linear pos of the pixel thread in block
	const uint threadPosInBlock = threadIdx.x + threadIdx.y * blockDim.x;

	uint offset = 0;

	while(offset < totalPixels)
	{		    
		int temp_x,temp_y;
		temp_x=startX + ((threadPosInBlock + offset) % width);
		temp_y=startY + ((threadPosInBlock + offset) / width);
		if(temp_x < 0)
			temp_x=0;
		if(temp_x >= iWidth)
			temp_x=iWidth-1;
		if(temp_y < 0)
			temp_y=0;
		if(temp_y >= iHeight)
			temp_y=iHeight-1;

		if(offset + threadPosInBlock < totalPixels)
			sdata3[offset + threadPosInBlock] = *((float3*)(((char*)inputImage) + temp_y*iPitchBytes) + temp_x);
		//	 sdata3[offset + threadPosInBlock] = inputImage[temp_x + temp_y * iPitchBytes];
		offset += blockPixels; 

	}

	__syncthreads();
		
	// convolution -  image on shared memory 

	const int kWidth  = (kRadiusX<<1) + 1;
	const int kHeight = (kRadiusY<<1) + 1;

	if(thposX < iWidth && thposY < iHeight)
	{
		float3 sum = make_float3(0.0f, 0.0f, 0.0f);

		for (int y = 0; y < kHeight; ++y)
		{
			for (int x = 0; x < kWidth; ++x) {
				const int sharedY = threadIdx.y + y;
				const int sharedX = threadIdx.x + x;

				sum.x += constKernel[y * kWidth + x] * sdata3[sharedX + sharedY * width].x;
				sum.y += constKernel[y * kWidth + x] * sdata3[sharedX + sharedY * width].y;
				sum.z += constKernel[y * kWidth + x] * sdata3[sharedX + sharedY * width].z;
//				sum += sdata[sharedY * width + sharedX] * constKernel[y * kWidth + x];
			}
		}
		*((float3*)(((char*)outputImage) + thposY*iPitchBytes)+ thposX) = sum;	
		
	}
	


} 




__global__ void gpu_ImageFloat3ToFloat4_d(const float3 *inputImage, float4 *outputImage, int iWidth, int iHeight, size_t iPitchBytes, size_t oPitchBytes)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= iWidth || y >= iHeight) return;

  float3 rgb = *((float3*)((char*)inputImage + y*iPitchBytes)+x);
  *((float4*)(((char*)outputImage) + y*oPitchBytes)+ x) = make_float4(rgb.x, rgb.y, rgb.z, 0.0f);
}





// mode 6 (interleaved): using texture memory for image and constant memory for kernel access
__global__ void gpu_convolutionInterleavedRGB_tex_cm_d(float3 *outputImage,
    int iWidth, int iHeight, int kRadiusX, int kRadiusY, size_t oPitchBytes)
{
	// ### implement me ### 
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= iWidth || y >= iHeight) return;

	const float xx = (float)x;
	const float yy = (float)y;
	const int kWidth = (kRadiusX<<1) + 1;

	int xk, yk;
	float3 value = make_float3(0.0f, 0.0f, 0.0f);

	float4 imageVal;
	for (xk=-kRadiusX; xk <= kRadiusX; xk++) 	  {
		for (yk=-kRadiusY; yk <= kRadiusY; yk++)   {
			imageVal = tex2D(tex_ImageF4,xx+xk,yy+yk);
			value.x += constKernel[(yk+kRadiusY)*kWidth + xk+kRadiusX] * imageVal.x;
			value.y += constKernel[(yk+kRadiusY)*kWidth + xk+kRadiusX] * imageVal.y;
			value.z += constKernel[(yk+kRadiusY)*kWidth + xk+kRadiusX] * imageVal.z;
		}
	}

	*((float3*)(((char*)outputImage) + y*oPitchBytes) + x) = value;
//	*((float3*)(((char*)outputImage) + y*oPitchBytes) + x) = make_float3(0,0,0);
}


void gpu_convolutionInterleavedRGB(const float *inputImage, const float *kernel, float *outputImage,
                                   int iWidth, int iHeight, int kRadiusX, int kRadiusY, int mode)
{
  size_t iPitchBytesF3, iPitchBytesF4;
  float3 *d_inputImageF3, *d_outputImageF3;
  float4 *d_inputImageF4;
  const int kWidth  = (kRadiusX<<1) + 1;
  const int kHeight = (kRadiusY<<1) + 1;

  //  allocate memory and copy data
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImageF3), &iPitchBytesF3, iWidth*sizeof(float3), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_outputImageF3), &iPitchBytesF3, iWidth*sizeof(float3), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImageF4), &iPitchBytesF4, iWidth*sizeof(float4), iHeight ) );

  cutilSafeCall( hipMemcpy2D(d_inputImageF3, iPitchBytesF3, inputImage, iWidth*sizeof(float3), iWidth*sizeof(float3), iHeight, hipMemcpyHostToDevice) );


  dim3 blockSize(BW,BH);  
  dim3 gridSize( ((iWidth%BW) ? (iWidth/BW+1) : (iWidth/BW)), ((iHeight%BH) ? (iHeight/BH+1) : (iHeight/BH)) );
  int smSizeF3 =  (blockSize.x+(kRadiusX<<1)) * (blockSize.y+(kRadiusY<<1)) * sizeof(float3);
  
  // convert image from float3* to float4*
  gpu_ImageFloat3ToFloat4_d<<<gridSize, blockSize>>>(d_inputImageF3, d_inputImageF4, iWidth, iHeight, iPitchBytesF3, iPitchBytesF4);
  
  gpu_bindConstantMemory(kernel, kWidth*kHeight);
  gpu_bindTextureMemoryF4(d_inputImageF4, iWidth, iHeight, iPitchBytesF4);


  switch(mode) {
    case 1:
    case 2:
    case 3:
    case 4:
      std::cout << "gpu_convolutionInterleavedRGB() Warning: mode " << mode << " is not supported." << std::endl;
      break;
    case 5:
      gpu_convolutionInterleavedRGB_dsm_cm_d<<<gridSize,blockSize,smSizeF3>>>(d_inputImageF3, d_outputImageF3,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
      break;
    case 6:
      gpu_convolutionInterleavedRGB_tex_cm_d<<<gridSize,blockSize>>>(d_outputImageF3,
        iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
      break;
    default:
      std::cout << "gpu_convolutionInterleavedRGB() Warning: mode " << mode << " is not supported." << std::endl;
  }

  cutilSafeCall( hipDeviceSynchronize() );
  cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float3), d_outputImageF3, iPitchBytesF3, iWidth*sizeof(float3), iHeight, hipMemcpyDeviceToHost) );


  // free memory
  gpu_unbindTextureMemoryF4();
  cutilSafeCall( hipFree(d_inputImageF4) );
  cutilSafeCall( hipFree(d_inputImageF3) );
  cutilSafeCall( hipFree(d_outputImageF3) );
}












//----------------------------------------------------------------------------
// Benchmark Functions
//----------------------------------------------------------------------------




void gpu_convolutionKernelBenchmarkGrayImage(const float *inputImage, const float *kernel, float *outputImage,
                                             int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                             int numKernelTestCalls)
{
  size_t iPitchBytes, kPitchBytes;
  size_t iPitch, kPitch;
  clock_t startTime, endTime;
  float *d_inputImage, *d_kernel, *d_outputImage;
  float fps;

  const int kWidth  = (kRadiusX<<1) + 1;
  const int kHeight = (kRadiusY<<1) + 1;

  assert(kWidth*kHeight <= MAXKERNELSIZE);

  dim3 blockSize(BW,BH);  
  dim3 gridSize( ((iWidth%BW) ? (iWidth/BW+1) : (iWidth/BW)), ((iHeight%BH) ? (iHeight/BH+1) : (iHeight/BH)) );
  int smSize =  (blockSize.x+(kRadiusX<<1)) * (blockSize.y+(kRadiusY<<1)) * sizeof(float);

  //  allocate memory and copy data
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImage), &iPitchBytes, iWidth*sizeof(float), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_outputImage), &iPitchBytes, iWidth*sizeof(float), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_kernel), &kPitchBytes, kWidth*sizeof(float), kHeight ) );   
  iPitch = iPitchBytes/sizeof(float);
  kPitch = kPitchBytes/sizeof(float);
  
  cutilSafeCall( hipMemcpy2D(d_inputImage, iPitchBytes, inputImage, iWidth*sizeof(float), iWidth*sizeof(float), iHeight, hipMemcpyHostToDevice) );
  cutilSafeCall( hipMemcpy2D(d_kernel, kPitchBytes, kernel, kWidth*sizeof(float), kWidth*sizeof(float), kHeight, hipMemcpyHostToDevice) );

  gpu_bindConstantMemory(kernel, kWidth*kHeight);
  gpu_bindTextureMemory(d_inputImage, iWidth, iHeight, iPitchBytes);

  // --- global memory only ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_gm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  fprintf(stderr, "%f fps - global memory only\n",fps);
  
  
  // --- global memory for image and constant memory for kernel access ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_gm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  fprintf(stderr, "%f fps - global memory for image & constant memory for kernel access\n",fps);


  // --- shared memory for image and global memory for kernel access ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_sm_d<<<gridSize,blockSize>>>(d_inputImage, d_kernel, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch, kPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  fprintf(stderr, "%f fps - shared memory for image & global memory for kernel access\n",fps);

  
  // --- shared memory for image and constant memory for kernel access ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_sm_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  fprintf(stderr, "%f fps - shared memory for image & constant memory for kernel access\n",fps);


   // --- shared memory for image and constant memory for kernel access ---  
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_dsm_cm_d<<<gridSize,blockSize,smSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  fprintf(stderr, "%f fps - dyn. shared memory for image & const memory for kernel access\n",fps);



  // --- texture memory for image and constant memory for kernel access ---
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionGrayImage_tex_cm_d<<<gridSize,blockSize>>>(d_inputImage, d_outputImage, iWidth, iHeight, kRadiusX, kRadiusY, iPitch);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC;
  fprintf(stderr, "%f fps - texture memory for image & const memory for kernel access\n",fps);


  cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float), d_outputImage, iPitchBytes, iWidth*sizeof(float), iHeight, hipMemcpyDeviceToHost) );

  // free memory
  gpu_unbindTextureMemory();
  cutilSafeCall( hipFree(d_inputImage) );
  cutilSafeCall( hipFree(d_outputImage) );
  cutilSafeCall( hipFree(d_kernel) );
}



void gpu_convolutionKernelBenchmarkInterleavedRGB(const float *inputImage, const float *kernel, float *outputImage,
                                                  int iWidth, int iHeight, int kRadiusX, int kRadiusY,
                                                  int numKernelTestCalls)
{
  size_t iPitchBytesF3, iPitchBytesF4;
  clock_t startTime, endTime;
  float3 *d_inputImageF3, *d_outputImageF3;
  float4 *d_inputImageF4;
  float fps;

  const int kWidth  = (kRadiusX<<1) + 1;
  const int kHeight = (kRadiusY<<1) + 1;

  assert(kWidth*kHeight <= MAXKERNELSIZE);

  dim3 blockSize(BW,BH);  
  dim3 gridSize( ((iWidth%BW) ? (iWidth/BW+1) : (iWidth/BW)), ((iHeight%BH) ? (iHeight/BH+1) : (iHeight/BH)) );
  int smSizeF3 =  (blockSize.x+(kRadiusX<<1)) * (blockSize.y+(kRadiusY<<1)) * sizeof(float3);

  //  allocate memory and copy data
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImageF3), &iPitchBytesF3, iWidth*sizeof(float3), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_inputImageF4), &iPitchBytesF4, iWidth*sizeof(float4), iHeight ) );
  cutilSafeCall( hipMallocPitch( (void**)&(d_outputImageF3), &iPitchBytesF3, iWidth*sizeof(float3), iHeight ) );   
  cutilSafeCall( hipMemcpy2D(d_inputImageF3, iPitchBytesF3, inputImage, iWidth*sizeof(float3), iWidth*sizeof(float3), iHeight, hipMemcpyHostToDevice) );

  gpu_bindConstantMemory(kernel, kWidth*kHeight);
  
  
  // --- shared memory for interleaved image and constant memory for kernel access ---  
  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionInterleavedRGB_dsm_cm_d<<<gridSize,blockSize,smSizeF3>>>(d_inputImageF3, d_outputImageF3, iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC * 3;
  fprintf(stderr, "%f fps - dyn. shared mem for interleaved img & const mem for kernel\n",fps);


  // --- texture memory for interleaved image and constant memory for kernel access ---
  gpu_ImageFloat3ToFloat4_d<<<gridSize, blockSize>>>(d_inputImageF3, d_inputImageF4, iWidth, iHeight, iPitchBytesF3, iPitchBytesF4);
  gpu_bindTextureMemoryF4(d_inputImageF4, iWidth, iHeight, iPitchBytesF4);

  startTime = clock();
  for(int c=0;c<numKernelTestCalls;c++) {
    gpu_convolutionInterleavedRGB_tex_cm_d<<<gridSize,blockSize>>>(d_outputImageF3, iWidth, iHeight, kRadiusX, kRadiusY, iPitchBytesF3);
    cutilSafeCall( hipDeviceSynchronize() );
  }
  endTime = clock();
  fps = (float)numKernelTestCalls / float(endTime - startTime) * CLOCKS_PER_SEC * 3;
  fprintf(stderr, "%f fps - texture mem for interleaved img & const mem for kernel access\n",fps);


  cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float3), d_outputImageF3, iPitchBytesF3, iWidth*sizeof(float3), iHeight, hipMemcpyDeviceToHost) );
  

  // free memory
  gpu_unbindTextureMemoryF4();
  cutilSafeCall( hipFree(d_inputImageF3) );
  cutilSafeCall( hipFree(d_outputImageF3) );
  cutilSafeCall( hipFree(d_inputImageF4) );
}





void gpu_bindConstantMemory(const float *kernel, int size) 
{
  cutilSafeCall( hipMemcpyToSymbol( (const char *)&constKernel, kernel, size*sizeof(float)) );
}



void gpu_bindTextureMemory(float *d_inputImage, int iWidth, int iHeight, size_t iPitchBytes)
{
  // >>>> prepare usage of texture memory
  tex_Image.addressMode[0] = hipAddressModeClamp;
  tex_Image.addressMode[1] = hipAddressModeClamp;
  tex_Image.filterMode = hipFilterModeLinear;
  tex_Image.normalized = false;
  // <<<< prepare usage of texture memory

  cutilSafeCall( hipBindTexture2D(0, &tex_Image, d_inputImage, &tex_Image_desc, iWidth, iHeight, iPitchBytes) );
}


void gpu_unbindTextureMemory()
{
  cutilSafeCall( hipUnbindTexture(tex_Image) );
}



void gpu_bindTextureMemoryF4(float4 *d_inputImageF4, int iWidth, int iHeight, size_t iPitchBytesF4)
{
  // >>>> prepare usage of texture memory
  tex_ImageF4.addressMode[0] = hipAddressModeClamp;
  tex_ImageF4.addressMode[1] = hipAddressModeClamp;
  tex_ImageF4.filterMode = hipFilterModeLinear;
  tex_ImageF4.normalized = false;
  // <<<< prepare usage of texture memory

  cutilSafeCall( hipBindTexture2D(0, &tex_ImageF4, d_inputImageF4, &tex_Image_descF4, iWidth, iHeight, iPitchBytesF4) );
}


void gpu_unbindTextureMemoryF4()
{
  cutilSafeCall( hipUnbindTexture(tex_ImageF4) );
}
