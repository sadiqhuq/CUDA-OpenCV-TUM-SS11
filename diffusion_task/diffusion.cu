#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    summer term 2011 / 19-26th September
*
* project: diffusion
* file:    diffusion.cu
*
* 
\********* PLEASE ENTER YOUR CORRECT STUDENT NAME AND ID BELOW **************/
const char* studentName = "Sadiq Huq";
const int   studentID   = 3273623;
/****************************************************************************\
*
* In this file the following methods have to be edited or completed:
*
* diffuse_linear_isotrop_shared(const float  *d_input, ... ) - Done
* diffuse_linear_isotrop_shared(const float3 *d_input, ... ) - Done
* diffuse_nonlinear_isotrop_shared(const float  *d_input, ... ) - Done
* diffuse_nonlinear_isotrop_shared(const float3 *d_input, ... ) - Done 
* compute_tv_diffusivity_shared - Done 
* compute_tv_diffusivity_joined_shared - Done 
* compute_tv_diffusivity_separate_shared - Done
* jacobi_shared(float  *d_output, ... ) - Done 
* jacobi_shared(float3 *d_output, ... ) - Done
* sor_shared(float  *d_output, ... ) - Done
* sor_shared(float3 *d_output, ... ) - Done
*
\****************************************************************************/


#define DIFF_BW 16
#define DIFF_BH 16

#define TV_EPSILON 0.1f



#include <cutil.h>
#include <cutil_inline.h>

#include "diffusion.cuh"



__host__ const char* getStudentName() { return studentName; };
__host__ int         getStudentID()   { return studentID; };
__host__ bool        checkStudentNameAndID() { return strcmp(studentName, "John Doe") != 0 && studentID != 1234567; };


//----------------------------------------------------------------------------
// Linear Diffusion
//----------------------------------------------------------------------------


__global__ void diffuse_linear_isotrop_shared(
  const float *d_input,
  float *d_output,
  float timeStep, 
  int nx, int ny,
  size_t pitch)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const int idx = y*pitch + x;

  __shared__ float u[DIFF_BW+2][DIFF_BH+2];

  // load data into shared memory
  if (x < nx && y < ny) {

	  u[tx][ty] = d_input[idx];

	  if (x == 0)  u[0][ty] = u[tx][ty];
	  else if (x == nx-1) u[tx+1][ty] = u[tx][ty];
	  else {
		  if (threadIdx.x == 0) u[0][ty] = d_input[idx-1];
		  else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = d_input[idx+1];
	  }

	  if (y == 0)  u[tx][0] = u[tx][ty];
	  else if (y == ny-1) u[tx][ty+1] = u[tx][ty];
	  else {
		  if (threadIdx.y == 0) u[tx][0] = d_input[idx-pitch];
		  else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = d_input[idx+pitch];
	  }
  }

  __syncthreads();

  // ### implement me ###
  
  if(x < nx && y < ny)
  {
	  float RHS = u[tx][ty] + ( timeStep * ( u[tx+1][ty] + u[tx-1][ty] + u[tx][ty+1] + u[tx][ty-1] - 4*u[tx][ty] ) )  ;
	  d_output[idx] = RHS;	  
  }

}


__global__ void diffuse_linear_isotrop_shared
(
 const float3 *d_input,
 float3 *d_output,
 float timeStep,
 int nx, int ny,
 size_t pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];
  float3 imgValue;

  // load data into shared memory
  if (x < nx && y < ny) {

    imgValue = *( (float3*)imgP );
    u[tx][ty] = imgValue;

    if (x == 0)  u[0][ty] = imgValue;
    else if (x == nx-1) u[tx+1][ty] = imgValue;
    else {
      if (threadIdx.x == 0) u[0][ty] = *( ((float3*)imgP)-1 );
      else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = *( ((float3*)imgP)+1 );
    }

    if (y == 0)  u[tx][0] = imgValue;
    else if (y == ny-1) u[tx][ty+1] = imgValue;
    else {
      if (threadIdx.y == 0) u[tx][0] = *( (float3*)(imgP-pitchBytes) );
      else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
    }
  }

  __syncthreads();
  
  // ### implement me ###
  
  float3 RHS3 = make_float3 (
		  u[tx][ty].x + ( timeStep * ( u[tx+1][ty].x + u[tx-1][ty].x + u[tx][ty+1].x + u[tx][ty-1].x - 4*u[tx][ty].x ) ),
		  u[tx][ty].y + ( timeStep * ( u[tx+1][ty].y + u[tx-1][ty].y + u[tx][ty+1].y + u[tx][ty-1].y - 4*u[tx][ty].y ) ),
		  u[tx][ty].z + ( timeStep * ( u[tx+1][ty].z + u[tx-1][ty].z + u[tx][ty+1].z + u[tx][ty-1].z - 4*u[tx][ty].z ) ) );

  *((float3*)(((char*)d_output) + y*pitchBytes)+ x) = RHS3 ;
}




//----------------------------------------------------------------------------
// Non-linear Diffusion - explicit scheme
//----------------------------------------------------------------------------




__global__ void diffuse_nonlinear_isotrop_shared
(
 const float *d_input,
 const float *d_diffusivity,
 float *d_output,
 float timeStep,
 int   nx,
 int   ny,
 size_t   pitch
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const int idx = y*pitch + x;

  __shared__ float u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float g[DIFF_BW+2][DIFF_BH+2];


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = d_input[idx];
    g[tx][ty] = d_diffusivity[idx];

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else {
      if (threadIdx.x == 0) {
        u[0][ty] = d_input[idx-1];
        g[0][ty] = d_diffusivity[idx-1];
      }
      else if (threadIdx.x == blockDim.x-1) {
        u[tx+1][ty] = d_input[idx+1];
        g[tx+1][ty] = d_diffusivity[idx+1];
      }
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else {
      if (threadIdx.y == 0) {
        u[tx][0] = d_input[idx-pitch];
        g[tx][0] = d_diffusivity[idx-pitch];
      }
      else if (threadIdx.y == blockDim.y-1) {
        u[tx][ty+1] = d_input[idx+pitch];
        g[tx][ty+1] = d_diffusivity[idx+pitch];
      }
    }
  }

  __syncthreads();
  
  // ### implement me ###
  
  if(x < nx && y < ny)
  {
	  float alpha = 0.5 * ( g[tx+1][ty] + g[tx][ty] );
	  float beta  = 0.5 * ( g[tx][ty] + g[tx-1][ty] );
	  float gamma = 0.5 * ( g[tx][ty+1] + g[tx][ty] );
	  float sigma = 0.5 * ( g[tx][ty] + g[tx][ty-1] );
	  
	  
	  float RHS = alpha * u[tx+1][ty] + beta * u[tx-1][ty] + gamma * u[tx][ty+1] + sigma * u[tx][ty-1] - (alpha+beta+gamma+sigma)*u[tx][ty] ;
	  d_output[idx] = u[tx][ty] + timeStep * RHS;	  
	  

  }
  
}




__global__ void diffuse_nonlinear_isotrop_shared
(
 const float3 *d_input,
 const float3 *d_diffusivity,
 float3 *d_output,
 float timeStep,
 int   nx,
 int   ny,
 size_t   pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);
  const char* diffP = (char*)d_diffusivity + y*pitchBytes + x*sizeof(float3);

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float3 g[DIFF_BW+2][DIFF_BH+2];


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = *( (float3*)imgP );
    g[tx][ty] = *( (float3*)diffP );

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else {
      if (threadIdx.x == 0) {
        u[0][ty] = *( ((float3*)imgP)-1 );
        g[0][ty] = *( ((float3*)diffP)-1 );
      }
      else if (threadIdx.x == blockDim.x-1) {
        u[tx+1][ty] = *( ((float3*)imgP)+1 );
        g[tx+1][ty] = *( ((float3*)diffP)+1 );
      }
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else {
      if (threadIdx.y == 0) {
        u[tx][0] = *( (float3*)(imgP-pitchBytes) );
        g[tx][0] = *( (float3*)(diffP-pitchBytes) );
      }
      else if (threadIdx.y == blockDim.y-1) {
        u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
        g[tx][ty+1] = *( (float3*)(diffP+pitchBytes) );
      }
    }
  }

  __syncthreads();

  
//   ### implement me ###
//
  if(x < nx && y < ny)
  {
	  float3 alpha = make_float3( 
			  0.5 * ( g[tx+1][ty].x + g[tx][ty].x),
			  0.5 * ( g[tx+1][ty].y + g[tx][ty].y),
			  0.5 * ( g[tx+1][ty].z + g[tx][ty].z) );

	  float3 beta  = make_float3( 
			  0.5 * ( g[tx][ty].x + g[tx-1][ty].x ),
			  0.5 * ( g[tx][ty].y + g[tx-1][ty].y ),
			  0.5 * ( g[tx][ty].z + g[tx-1][ty].z ));
	  
	  float3 gamma = make_float3(
			  0.5 * ( g[tx][ty+1].x + g[tx][ty].x ),
			  0.5 * ( g[tx][ty+1].y + g[tx][ty].y ),
			  0.5 * ( g[tx][ty+1].z + g[tx][ty].z ) );
	  
	  float3 sigma = make_float3(
			  0.5 * ( g[tx][ty].x + g[tx][ty-1].x ),
			  0.5 * ( g[tx][ty].y + g[tx][ty-1].y ),
			  0.5 * ( g[tx][ty].z + g[tx][ty-1].z ) );
	  
//	  float3 RHS3 = alpha * u[tx+1][ty] + beta * u[tx-1][ty] + gamma * u[tx][ty+1] + sigma * u[tx][ty-1] - (alpha+beta+gamma+sigma)*u[tx][ty] ;
//	  d_output[idx] = u[tx][ty] + timeStep * RHS;	  
	
	    float3 RHS3 = make_float3 (
	  		  alpha.x * u[tx+1][ty].x + beta.x * u[tx-1][ty].x + gamma.x * u[tx][ty+1].x + sigma.x * u[tx][ty-1].x - (alpha.x+beta.x+gamma.x+sigma.x)*u[tx][ty].x,
	  		  alpha.y * u[tx+1][ty].y + beta.y * u[tx-1][ty].y + gamma.y * u[tx][ty+1].y + sigma.y * u[tx][ty-1].y - (alpha.y+beta.y+gamma.y+sigma.y)*u[tx][ty].y,
	  		  alpha.z * u[tx+1][ty].z + beta.z * u[tx-1][ty].z + gamma.z * u[tx][ty+1].z + sigma.z * u[tx][ty-1].z - (alpha.z+beta.z+gamma.z+sigma.z)*u[tx][ty].z);
	  
	     *((float3*)(((char*)d_output) + y*pitchBytes)+ x) =  make_float3 ( 
	    		 u[tx][ty].x + timeStep * RHS3.x ,
	    		 u[tx][ty].y + timeStep * RHS3.y,
	    		 u[tx][ty].z + timeStep * RHS3.z );


  
  }
 

}



__global__ void compute_tv_diffusivity_shared
(
 const float *d_input,
 float *d_output,
 int   nx,
 int   ny,
 size_t   pitch
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const int idx = y*pitch + x;

  __shared__ float u[DIFF_BW+2][DIFF_BH+2];

  // load data into shared memory
  if (x < nx && y < ny) {

    u[tx][ty] = d_input[idx];

    if (x == 0)  u[0][ty] = u[tx][ty];
    else if (x == nx-1) u[tx+1][ty] = u[tx][ty];
    else {
      if (threadIdx.x == 0) u[0][ty] = d_input[idx-1];
      else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = d_input[idx+1];
    }

    if (y == 0)  u[tx][0] = u[tx][ty];
    else if (y == ny-1) u[tx][ty+1] = u[tx][ty];
    else {
      if (threadIdx.y == 0) u[tx][0] = d_input[idx-pitch];
      else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = d_input[idx+pitch];
    }
  }

  __syncthreads();

  
  // make use of the constant TV_EPSILON

  // ### implement me ###
  
  
  if(x < nx && y < ny)
  {
    float dx = (u[tx + 1][ty] - u[tx - 1][ty]);
    float dy = (u[tx][ty + 1] - u[tx][ty - 1]);

    d_output[idx] = 1.0f / sqrtf(dx * dx + dy * dy + TV_EPSILON);
  }
  

}



__global__ void compute_tv_diffusivity_joined_shared
(
 const float3 *d_input,
 float3 *d_output,
 int   nx,
 int   ny,
 size_t   pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];

  // load data into shared memory
  if (x < nx && y < ny) {

    u[tx][ty] = *( (float3*)imgP );

    if (x == 0)  u[0][ty] = u[tx][ty];
    else if (x == nx-1) u[tx+1][ty] = u[tx][ty];
    else {
      if (threadIdx.x == 0) u[0][ty] = *( ((float3*)imgP)-1 );
      else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = *( ((float3*)imgP)+1 );
    }

    if (y == 0)  u[tx][0] = u[tx][ty];
    else if (y == ny-1) u[tx][ty+1] = u[tx][ty];
    else {
      if (threadIdx.y == 0) u[tx][0] = *( (float3*)(imgP-pitchBytes) );
      else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
    }
  }

  __syncthreads();
  
  // make use of the constant TV_EPSILON

  // ### implement me ###
  
  if(x < nx && y < ny)
    {
	  float3 dx = make_float3 (
			  0.5*(u[tx + 1][ty].x - u[tx - 1][ty].x),
			  0.5*(u[tx + 1][ty].y - u[tx - 1][ty].y),
			  0.5*(u[tx + 1][ty].z - u[tx - 1][ty].z) );
	  
	  float3 dy = make_float3(
			  0.5*(u[tx][ty + 1].x - u[tx][ty - 1].x),
			  0.5*(u[tx][ty + 1].y - u[tx][ty - 1].y),
			  0.5*(u[tx][ty + 1].z - u[tx][ty - 1].z) );


      float3 RHS3 = make_float3 (
    		  1.0f / sqrtf(dx.x * dx.x + dy.x * dy.x + TV_EPSILON),
    		  1.0f / sqrtf(dx.y * dx.y + dy.y * dy.y + TV_EPSILON),
    		  1.0f / sqrtf(dx.z * dx.z + dy.z * dy.z + TV_EPSILON) );
      
      const float diffuse = ( (RHS3.x+RHS3.y+RHS3.z)/3 );
      *((float3*)(((char*)d_output) + y*pitchBytes)+ x) = make_float3(10,10,10) ; // Debug
      *((float3*)(((char*)d_output) + y*pitchBytes)+ x) = make_float3(diffuse,diffuse,diffuse) ;
      

    }
    
    
  
}



__global__ void compute_tv_diffusivity_separate_shared
(
 const float3 *d_input,
 float3 *d_output,
 int   nx,
 int   ny,
 size_t   pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];

  // load data into shared memory
  if (x < nx && y < ny) {

    u[tx][ty] = *( (float3*)imgP );

    if (x == 0)  u[threadIdx.x][ty] = u[tx][ty];
    else if (x == nx-1) u[tx+1][ty] = u[tx][ty];
    else {
      if (threadIdx.x == 0) u[0][ty] = *( ((float3*)imgP)-1 );
      else if (threadIdx.x == blockDim.x-1) u[tx+1][ty] = *( ((float3*)imgP)+1 );
    }

    if (y == 0)  u[tx][0] = u[tx][ty];
    else if (y == ny-1) u[tx][ty+1] = u[tx][ty];
    else {
      if (threadIdx.y == 0) u[tx][0] = *( (float3*)(imgP-pitchBytes) );
      else if (threadIdx.y == blockDim.y-1) u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
    }
  }

  __syncthreads();


  // make use of the constant TV_EPSILON
 

  if(x < nx && y < ny)
  {
	  float3 result;
	  float dx = 0.5f*(u[tx+1][ty].x-u[tx-1][ty].x);
	  float dy = 0.5f*(u[tx][ty+1].x-u[tx][ty-1].x);
	  result.x = 1.0f / (sqrt(dx*dx+dy*dy+TV_EPSILON));

	  dx = 0.5f*(u[tx+1][ty].y-u[tx-1][ty].y);
	  dy = 0.5f*(u[tx][ty+1].y-u[tx][ty-1].y);
	  result.y = 1.0f / (sqrt(dx*dx+dy*dy+TV_EPSILON));

	  dx = 0.5f*(u[tx+1][ty].z-u[tx-1][ty].z);
	  dy = 0.5f*(u[tx][ty+1].z-u[tx][ty-1].z);
	  result.z = 1.0f / (sqrt(dx*dx+dy*dy+TV_EPSILON));

//	  *((float3*)(((char*)d_output) + y*pitchBytes)+ x) = make_float3(10,10,10) ; // Debug
	  *((float3*)((char*)d_output + y*pitchBytes + x*sizeof(float3))) =result;
	  
  }
}




//----------------------------------------------------------------------------
// Non-linear Diffusion - Jacobi scheme
//----------------------------------------------------------------------------



__global__ void jacobi_shared
(
 float *d_output,
 const float *d_input,
 const float *d_original,
 const float *d_diffusivity,
 float weight,
 float overrelaxation,
 int   nx,
 int   ny,
 size_t   pitch
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int idx = y*pitch + x;

  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;

  __shared__ float u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float g[DIFF_BW+2][DIFF_BH+2];


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = d_input[idx];
    g[tx][ty] = d_diffusivity[idx];

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else {
      if (threadIdx.x == 0) {
        u[0][ty] = d_input[idx-1];
        g[0][ty] = d_diffusivity[idx-1];
      }
      else if (threadIdx.x == blockDim.x-1) {
        u[tx+1][ty] = d_input[idx+1];
        g[tx+1][ty] = d_diffusivity[idx+1];
      }
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else {
      if (threadIdx.y == 0) {
        u[tx][0] = d_input[idx-pitch];
        g[tx][0] = d_diffusivity[idx-pitch];
      }
      else if (threadIdx.y == blockDim.y-1) {
        u[tx][ty+1] = d_input[idx+pitch];
        g[tx][ty+1] = d_diffusivity[idx+pitch];
      }
    }
  }

  __syncthreads();
 


  // ### implement me ###

  
  if (x < nx && y < ny){
	  
	  
	  float alpha = 0.5 * ( g[tx+1][ty] + g[tx][ty]   );
	  float beta  = 0.5 * ( g[tx][ty]   + g[tx-1][ty] );
	  float gamma = 0.5 * ( g[tx][ty+1] + g[tx][ty]   );
	  float sigma = 0.5 * ( g[tx][ty]   + g[tx][ty-1] );

	  // B.C
	  if (x >= nx) alpha = 0;
	  if (x <= 0)  beta = 0;
	  if (y >= ny) gamma = 0;
	  if (y <= 0)  sigma = 0;

	  float RHS = d_original[idx]+ weight * (alpha * u[tx+1][ty] + beta * u[tx-1][ty] + gamma * u[tx][ty+1] + sigma * u[tx][ty-1] );

	  d_output[idx] = RHS /(1 + weight * (alpha + beta + gamma + sigma));
  }

}



__global__ void jacobi_shared
(
 float3 *d_output,
 const float3 *d_input,
 const float3 *d_original,
 const float3 *d_diffusivity,
 float weight,
 float overrelaxation,
 int   nx,
 int   ny,
 size_t   pitchBytes
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);
  const char* diffP = (char*)d_diffusivity + y*pitchBytes + x*sizeof(float3);

  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float3 g[DIFF_BW+2][DIFF_BH+2];


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = *( (float3*)imgP );
    g[tx][ty] = *( (float3*)diffP );

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else {
      if (threadIdx.x == 0) {
        u[0][ty] = *( ((float3*)imgP)-1 );
        g[0][ty] = *( ((float3*)diffP)-1 );
      }
      else if (threadIdx.x == blockDim.x-1) {
        u[tx+1][ty] = *( ((float3*)imgP)+1 );
        g[tx+1][ty] = *( ((float3*)diffP)+1 );
      }
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else {
      if (threadIdx.y == 0) {
        u[tx][0] = *( (float3*)(imgP-pitchBytes) );
        g[tx][0] = *( (float3*)(diffP-pitchBytes) );
      }
      else if (threadIdx.y == blockDim.y-1) {
        u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
        g[tx][ty+1] = *( (float3*)(diffP+pitchBytes) );
      }
    }
  }

  __syncthreads();


  // ### implement me ###

  if (x < nx && y < ny){


	  float3 alpha = make_float3( 
			  0.5 * ( g[tx+1][ty].x + g[tx][ty].x),
			  0.5 * ( g[tx+1][ty].y + g[tx][ty].y),
			  0.5 * ( g[tx+1][ty].z + g[tx][ty].z) );

	  float3 beta  = make_float3( 
			  0.5 * ( g[tx][ty].x + g[tx-1][ty].x ),
			  0.5 * ( g[tx][ty].y + g[tx-1][ty].y ),
			  0.5 * ( g[tx][ty].z + g[tx-1][ty].z ));

	  float3 gamma = make_float3(
			  0.5 * ( g[tx][ty+1].x + g[tx][ty].x ),
			  0.5 * ( g[tx][ty+1].y + g[tx][ty].y ),
			  0.5 * ( g[tx][ty+1].z + g[tx][ty].z ) );

	  float3 sigma = make_float3(
			  0.5 * ( g[tx][ty].x + g[tx][ty-1].x ),
			  0.5 * ( g[tx][ty].y + g[tx][ty-1].y ),
			  0.5 * ( g[tx][ty].z + g[tx][ty-1].z ) );


	  if (x >= nx) alpha = make_float3(0,0,0);
	  if (x <= 0)  beta  = make_float3(0,0,0);
	  if (y >= ny) gamma = make_float3(0,0,0);
	  if (y <= 0)  sigma = make_float3(0,0,0);

	  float3 Img = *((float3*)(((char*)d_original) + y*pitchBytes)+ x) ;

	  float3 RHS3 = make_float3 (
			  alpha.x * u[tx+1][ty].x + beta.x * u[tx-1][ty].x + gamma.x * u[tx][ty+1].x + sigma.x * u[tx][ty-1].x ,
			  alpha.y * u[tx+1][ty].y + beta.y * u[tx-1][ty].y + gamma.y * u[tx][ty+1].y + sigma.y * u[tx][ty-1].y ,
			  alpha.z * u[tx+1][ty].z + beta.z * u[tx-1][ty].z + gamma.z * u[tx][ty+1].z + sigma.z * u[tx][ty-1].z );

	  float3 RHSdenom = make_float3(
			  1 + weight * (alpha.x + beta.x+gamma.x+sigma.x),
			  1 + weight * (alpha.y + beta.y+gamma.y+sigma.y),
			  1 + weight * (alpha.z + beta.z+gamma.z+sigma.z) );

	  *((float3*)(((char*)d_output) + y*pitchBytes)+ x) = make_float3(
			  (Img.x + weight * RHS3.x )/ RHSdenom.x,	  
			  (Img.y + weight * RHS3.y )/ RHSdenom.y,
			  (Img.z + weight * RHS3.z )/ RHSdenom.z) ; 

//	  	  	*((float3*)(((char*)d_output) + y*pitchBytes)+ x) = make_float3 (0,255,255); // Debug

  }
    
}



//----------------------------------------------------------------------------
// Non-linear Diffusion - Successive Over-Relaxation (SOR)
//----------------------------------------------------------------------------


__global__ void sor_shared
(
 float *d_output,
 const float *d_input,
 const float *d_original,
 const float *d_diffusivity,
 float weight,
 float overrelaxation,
 int   nx,
 int   ny,
 size_t   pitch,
 int   red
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int idx = y*pitch + x;
  
  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;

  __shared__ float u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float g[DIFF_BW+2][DIFF_BH+2];


  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = d_input[idx];
    g[tx][ty] = d_diffusivity[idx];

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else {
      if (threadIdx.x == 0) {
        u[0][ty] = d_input[idx-1];
        g[0][ty] = d_diffusivity[idx-1];
      }
      else if (threadIdx.x == blockDim.x-1) {
        u[tx+1][ty] = d_input[idx+1];
        g[tx+1][ty] = d_diffusivity[idx+1];
      }
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else {
      if (threadIdx.y == 0) {
        u[tx][0] = d_input[idx-pitch];
        g[tx][0] = d_diffusivity[idx-pitch];
      }
      else if (threadIdx.y == blockDim.y-1) {
        u[tx][ty+1] = d_input[idx+pitch];
        g[tx][ty+1] = d_diffusivity[idx+pitch];
      }
    }
  }

  __syncthreads();


  // ### implement me ###
  
  if ( (x+y)%2 == red )
  {
	  if (x < nx && y < ny){


		  float alpha = 0.5 * ( g[tx+1][ty] + g[tx][ty]   );
		  float beta  = 0.5 * ( g[tx][ty]   + g[tx-1][ty] );
		  float gamma = 0.5 * ( g[tx][ty+1] + g[tx][ty]   );
		  float sigma = 0.5 * ( g[tx][ty]   + g[tx][ty-1] );

		  // B.C
		  if (x >= nx) alpha = 0;
		  if (x <= 0)  beta = 0;
		  if (y >= ny) gamma = 0;
		  if (y <= 0)  sigma = 0;

		  float RHS = d_original[idx]+ weight * (alpha * u[tx+1][ty] + beta * u[tx-1][ty] + gamma * u[tx][ty+1] + sigma * u[tx][ty-1] );

		  //	  d_output[idx] = RHS /(1 + weight * (alpha + beta + gamma + sigma));


		  d_output[idx]  = (1 - overrelaxation ) * u[tx][ty] 
		                                                 + overrelaxation *  RHS / (1 + weight * (alpha + beta + gamma + sigma));


		  //	  d_output[idx] = RHS;  // Debug 
	  }
  }


}




__global__ void sor_shared
(
 float3 *d_output,
 const float3 *d_input,
 const float3 *d_original,
 const float3 *d_diffusivity,
 float weight,
 float overrelaxation,
 int   nx,
 int   ny,
 size_t   pitchBytes,
 int   red
 )
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const char* imgP = (char*)d_input + y*pitchBytes + x*sizeof(float3);
  const char* diffP = (char*)d_diffusivity + y*pitchBytes + x*sizeof(float3);

  const int tx = threadIdx.x+1;
  const int ty = threadIdx.y+1;

  __shared__ float3 u[DIFF_BW+2][DIFF_BH+2];
  __shared__ float3 g[DIFF_BW+2][DIFF_BH+2];



  // load data into shared memory
  if (x < nx && y < ny) {
    u[tx][ty] = *( (float3*)imgP );
    g[tx][ty] = *( (float3*)diffP );

    if (x == 0)  {
      u[0][ty] = u[tx][ty];
      g[0][ty] = g[tx][ty];
    }
    else if (x == nx-1) {
      u[tx+1][ty] = u[tx][ty];
      g[tx+1][ty] = g[tx][ty];
    }
    else {
      if (threadIdx.x == 0) {
        u[0][ty] = *( ((float3*)imgP)-1 );
        g[0][ty] = *( ((float3*)diffP)-1 );
      }
      else if (threadIdx.x == blockDim.x-1) {
        u[tx+1][ty] = *( ((float3*)imgP)+1 );
        g[tx+1][ty] = *( ((float3*)diffP)+1 );
      }
    }

    if (y == 0) {
      u[tx][0] = u[tx][ty];
      g[tx][0] = g[tx][ty];
    }
    else if (y == ny-1) {
      u[tx][ty+1] = u[tx][ty];
      g[tx][ty+1] = g[tx][ty];
    }
    else {
      if (threadIdx.y == 0) {
        u[tx][0] = *( (float3*)(imgP-pitchBytes) );
        g[tx][0] = *( (float3*)(diffP-pitchBytes) );
      }
      else if (threadIdx.y == blockDim.y-1) {
        u[tx][ty+1] = *( (float3*)(imgP+pitchBytes) );
        g[tx][ty+1] = *( (float3*)(diffP+pitchBytes) );
      }
    }
  }

  __syncthreads();


  // ### implement me ###
  if ( (x+y)%2== red )
  {
	  if (x < nx && y < ny){


		  float3 alpha = make_float3( 
				  0.5 * ( g[tx+1][ty].x + g[tx][ty].x),
				  0.5 * ( g[tx+1][ty].y + g[tx][ty].y),
				  0.5 * ( g[tx+1][ty].z + g[tx][ty].z) );

		  float3 beta  = make_float3( 
				  0.5 * ( g[tx][ty].x + g[tx-1][ty].x ),
				  0.5 * ( g[tx][ty].y + g[tx-1][ty].y ),
				  0.5 * ( g[tx][ty].z + g[tx-1][ty].z ));

		  float3 gamma = make_float3(
				  0.5 * ( g[tx][ty+1].x + g[tx][ty].x ),
				  0.5 * ( g[tx][ty+1].y + g[tx][ty].y ),
				  0.5 * ( g[tx][ty+1].z + g[tx][ty].z ) );

		  float3 sigma = make_float3(
				  0.5 * ( g[tx][ty].x + g[tx][ty-1].x ),
				  0.5 * ( g[tx][ty].y + g[tx][ty-1].y ),
				  0.5 * ( g[tx][ty].z + g[tx][ty-1].z ) );


		  if (x >= nx) alpha = make_float3(0,0,0);
		  if (x <= 0)  beta  = make_float3(0,0,0);
		  if (y >= ny) gamma = make_float3(0,0,0);
		  if (y <= 0)  sigma = make_float3(0,0,0);

		  float3 Img = *((float3*)(((char*)d_original) + y*pitchBytes)+ x) ;

		  float3 RHS3 = make_float3 (
				  alpha.x * u[tx+1][ty].x + beta.x * u[tx-1][ty].x + gamma.x * u[tx][ty+1].x + sigma.x * u[tx][ty-1].x ,
				  alpha.y * u[tx+1][ty].y + beta.y * u[tx-1][ty].y + gamma.y * u[tx][ty+1].y + sigma.y * u[tx][ty-1].y ,
				  alpha.z * u[tx+1][ty].z + beta.z * u[tx-1][ty].z + gamma.z * u[tx][ty+1].z + sigma.z * u[tx][ty-1].z );

		  float3 RHSdenom = make_float3(
				  1 + weight * (alpha.x + beta.x+gamma.x+sigma.x),
				  1 + weight * (alpha.y + beta.y+gamma.y+sigma.y),
				  1 + weight * (alpha.z + beta.z+gamma.z+sigma.z) );

		  float3 xbar = make_float3(
				  (Img.x + weight * RHS3.x )/ RHSdenom.x,	  
				  (Img.y + weight * RHS3.y )/ RHSdenom.y,
				  (Img.z + weight * RHS3.z )/ RHSdenom.z) ; 

		  //	  	  	*((float3*)(((char*)d_output) + y*pitchBytes)+ x) = make_float3 (0,255,255); // Debug
		  //	  d_output[idx]  = (1 - overrelaxation ) * u[tx][ty] 
		  //	          + overrelaxation *  RHS / (1 + weight * (alpha + beta + gamma + sigma));
		  *((float3*)(((char*)d_output) + y*pitchBytes)+ x) = make_float3 (
		  (1 - overrelaxation )*u[tx][ty].x + overrelaxation * xbar.x,
		  (1 - overrelaxation )*u[tx][ty].y + overrelaxation * xbar.y,
		  (1 - overrelaxation )*u[tx][ty].z + overrelaxation * xbar.z   );
		  
//		  *((float3*)(((char*)d_output) + y*pitchBytes)+ x) = make_float3(0,0,0);
	  }
  }
  
  
  
}




//----------------------------------------------------------------------------
// Host function
//----------------------------------------------------------------------------



void gpu_diffusion
(
 const float *input,
 float *output,
 int nx, int ny, int nc, 
 float timeStep,
 int iterations,
 float weight,
 int lagged_iterations,
 float overrelaxation,
 int mode
 )
{
  int i,j;
  size_t pitchF1, pitchBytesF1, pitchBytesF3;
  float *d_input = 0;
  float *d_output = 0;
  float *d_diffusivity = 0;
  float *d_original = 0;
  float *temp = 0;

  dim3 dimGrid((int)ceil((float)nx/DIFF_BW), (int)ceil((float)ny/DIFF_BH));
  dim3 dimBlock(DIFF_BW,DIFF_BH);

  // Allocation of GPU Memory
  if (nc == 1) {

    cutilSafeCall( hipMallocPitch( (void**)&(d_input), &pitchBytesF1, nx*sizeof(float), ny ) );
    cutilSafeCall( hipMallocPitch( (void**)&(d_output), &pitchBytesF1, nx*sizeof(float), ny ) );
    if (mode) cutilSafeCall( hipMallocPitch( (void**)&(d_diffusivity), &pitchBytesF1, nx*sizeof(float), ny ) );
    if (mode >= 2) cutilSafeCall( hipMallocPitch( (void**)&(d_original), &pitchBytesF1, nx*sizeof(float), ny ) );

    cutilSafeCall( hipMemcpy2D(d_input, pitchBytesF1, input, nx*sizeof(float), nx*sizeof(float), ny, hipMemcpyHostToDevice) );
    if (mode >= 2) cutilSafeCall( hipMemcpy2D(d_original, pitchBytesF1, d_input, pitchBytesF1, nx*sizeof(float), ny, hipMemcpyDeviceToDevice) );

    pitchF1 = pitchBytesF1/sizeof(float);

  } else if (nc == 3) {

    cutilSafeCall( hipMallocPitch( (void**)&(d_input), &pitchBytesF3, nx*sizeof(float3), ny ) );
    cutilSafeCall( hipMallocPitch( (void**)&(d_output), &pitchBytesF3, nx*sizeof(float3), ny ) );
    if (mode) cutilSafeCall( hipMallocPitch( (void**)&(d_diffusivity), &pitchBytesF3, nx*sizeof(float3), ny ) );
    if (mode >= 2) cutilSafeCall( hipMallocPitch( (void**)&(d_original), &pitchBytesF3, nx*sizeof(float3), ny ) );

    cutilSafeCall( hipMemcpy2D(d_input, pitchBytesF3, input, nx*sizeof(float3), nx*sizeof(float3), ny, hipMemcpyHostToDevice) );
    if (mode >= 2) cutilSafeCall( hipMemcpy2D(d_original, pitchBytesF3, d_input, pitchBytesF3, nx*sizeof(float3), ny, hipMemcpyDeviceToDevice) );

  }


  //Execution of the Diffusion Kernel

  if (mode == 0) {   // linear isotropic diffision
    if (nc == 1) {
      for (i=0;i<iterations;i++) {
        diffuse_linear_isotrop_shared<<<dimGrid,dimBlock>>>(d_input, d_output, timeStep, nx, ny, pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        temp = d_input;
        d_input = d_output;
        d_output = temp;
      }
    }
    else if (nc == 3) {
      for (i=0;i<iterations;i++) {
        diffuse_linear_isotrop_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_output,timeStep,nx,ny,pitchBytesF3);

        cutilSafeCall( hipDeviceSynchronize() );

        temp = d_input;
        d_input = d_output;
        d_output = temp;
      }
    }
  }
  else if (mode == 1) {  // nonlinear isotropic diffusion
    if (nc == 1) {

      for (i=0;i<iterations;i++) {
        compute_tv_diffusivity_shared<<<dimGrid,dimBlock>>>(d_input,d_diffusivity,nx,ny,pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        diffuse_nonlinear_isotrop_shared<<<dimGrid,dimBlock>>>(d_input,d_diffusivity,d_output,timeStep,nx,ny,pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        temp = d_input;
        d_input = d_output;
        d_output = temp;
      }
    }
    else if (nc == 3) {
      for (i=0;i<iterations;i++) {
    	  
    	  // Added 
//        compute_tv_diffusivity_separate_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);
    	  compute_tv_diffusivity_joined_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);

        cutilSafeCall( hipDeviceSynchronize() );

        diffuse_nonlinear_isotrop_shared<<<dimGrid,dimBlock>>>
          ((float3*)d_input,(float3*)d_diffusivity,(float3*)d_output,timeStep,nx,ny,pitchBytesF3);

        cutilSafeCall( hipDeviceSynchronize() );

        temp = d_input;
        d_input = d_output;
        d_output = temp;
      }
    }
  }
  else if (mode == 2) {    // Jacobi-method
    if (nc == 1) {
      for (i=0;i<iterations;i++) {
        compute_tv_diffusivity_shared<<<dimGrid,dimBlock>>>(d_input,d_diffusivity,nx,ny,pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        for (j=0;j<lagged_iterations;j++) {
          jacobi_shared<<<dimGrid,dimBlock>>> (d_output,d_input,d_original,
            d_diffusivity,weight,overrelaxation,nx,ny,pitchF1);

          cutilSafeCall( hipDeviceSynchronize() );

          temp = d_input;
          d_input = d_output;
          d_output = temp;
        }
      }
    }
    else if (nc == 3) {
      for (i=0;i<iterations;i++) {
        //--- this doesn't work with joint diffusivities ---
        //compute_tv_diffusivity_joined_shared<<<dimGrid,dimBlock>>>
        //		((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitch);
        compute_tv_diffusivity_separate_shared<<<dimGrid,dimBlock>>>
          ((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);

        cutilSafeCall( hipDeviceSynchronize() );

        for (j=0;j<lagged_iterations;j++) {
          jacobi_shared<<<dimGrid,dimBlock>>>
            ((float3*)d_output,(float3*)d_input,
            (float3*)d_original,(float3*)d_diffusivity,
            weight,overrelaxation,nx,ny,pitchBytesF3);

          cutilSafeCall( hipDeviceSynchronize() );

          temp = d_input;
          d_input = d_output;
          d_output = temp;
        }
      }
    }    
  }
  else if(mode == 3) {    // Successive Over Relaxation (Gauss-Seidel with extrapolation)
    if (nc == 1) {
      for (i=0;i<iterations;i++) {
        compute_tv_diffusivity_shared<<<dimGrid,dimBlock>>>(d_input,d_diffusivity,nx,ny,pitchF1);

        cutilSafeCall( hipDeviceSynchronize() );

        for(j=0;j<lagged_iterations;j++) {					
          sor_shared<<<dimGrid,dimBlock>>>(d_input,d_input,d_original,
            d_diffusivity,weight,overrelaxation,nx,ny,pitchF1, 0);

          cutilSafeCall( hipDeviceSynchronize() );

          sor_shared<<<dimGrid,dimBlock>>>(d_input,d_input,d_original,
            d_diffusivity,weight,overrelaxation,nx,ny,pitchF1, 1);

          cutilSafeCall( hipDeviceSynchronize() );
        }
      }
    }
    if (nc == 3) {
      for (i=0;i<iterations;i++) {
        compute_tv_diffusivity_separate_shared<<<dimGrid,dimBlock>>>((float3*)d_input,(float3*)d_diffusivity,nx,ny,pitchBytesF3);

        cutilSafeCall( hipDeviceSynchronize() );

        for (j=0;j<lagged_iterations;j++) {
          sor_shared<<<dimGrid,dimBlock>>>
            ((float3*)d_input,(float3*)d_input,
            (float3*)d_original,(float3*)d_diffusivity,
            weight,overrelaxation,nx,ny,pitchBytesF3, 0);

          cutilSafeCall( hipDeviceSynchronize() );

          sor_shared<<<dimGrid,dimBlock>>>
            ((float3*)d_input,(float3*)d_input,
            (float3*)d_original,(float3*)d_diffusivity,
            weight,overrelaxation,nx,ny,pitchBytesF3, 1);

          cutilSafeCall( hipDeviceSynchronize() );
        }
      }
    }
  }


  if (nc == 1) {
    if (mode == 3) cutilSafeCall( hipMemcpy2D(output, nx*sizeof(float), d_input, pitchBytesF1, nx*sizeof(float), ny, hipMemcpyDeviceToHost) );
    else cutilSafeCall( hipMemcpy2D(output, nx*sizeof(float), d_output, pitchBytesF1, nx*sizeof(float), ny, hipMemcpyDeviceToHost) );
  } else if (nc == 3) {
    if (mode == 3) cutilSafeCall( hipMemcpy2D(output, nx*sizeof(float3), d_input, pitchBytesF3, nx*sizeof(float3), ny, hipMemcpyDeviceToHost) );
    else cutilSafeCall( hipMemcpy2D(output, nx*sizeof(float3), d_output, pitchBytesF3, nx*sizeof(float3), ny, hipMemcpyDeviceToHost) );
  }


  // clean up
  if (d_original) cutilSafeCall( hipFree(d_original) );
  if (d_diffusivity) cutilSafeCall( hipFree(d_diffusivity) );
  if (d_output) cutilSafeCall( hipFree(d_output) );
  if (d_input)  cutilSafeCall( hipFree(d_input) );
}
