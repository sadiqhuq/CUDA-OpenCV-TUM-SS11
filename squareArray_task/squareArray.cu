/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    summer term 2011 / 19-26th September
*
* project: squareArray
* file:    squareArray.cu
*
*
* In this file the following methods have to be edited or completed:
*
* square_array_kernel
* square_array_gpu
* 
\****************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>


void square_array_cpu(float *a, unsigned int numElements)
{
  for (int i=0; i<numElements; i++)
    a[i] = a[i]*a[i];
}


// Kernel that executes on the CUDA device
__global__ void square_array_kernel(float *a, unsigned int numElements)
{
  // kernel code
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	  if (i < numElements ){
	    a[i] = a[i]*a[i]; 
	  }

}


// function that invokes the gpu kernel
__host__ void square_array_gpu(float *a_host, unsigned int numElements)
{
  float *a_device;
  size_t size = numElements*sizeof(float);

  // allocate memory on the device
  hipMalloc((void **) &a_device, size); 

  // copy array from host to device memory
  hipMemcpy(a_device, a_host, size, hipMemcpyHostToDevice);

  // do calculation on device
  int block_size = 4;
  int grid_size = numElements/block_size + (numElements%block_size ? 1:0);
  
    square_array_kernel <<< grid_size, block_size>>> (a_device, numElements);
  
  // Retrieve result from device and store it in host array
  hipMemcpy(a_host, a_device, size, hipMemcpyDeviceToHost);

  // free device memory
  hipFree(a_device);
}



// main routine that executes on the host
int main(void)
{
  float *a_host;                            // pointer to array in host memory
  const unsigned int numElements = 10;      // number of elements in the array
  size_t size = numElements * sizeof(float);
  a_host = (float *)malloc(size);           // allocate array on host
  
  // initialize host array with some data
  for (int i=0; i<numElements; i++) a_host[i] = (float)i;
  printf("\nCPU-version:\n");

  square_array_cpu(a_host, numElements);
  
  // print results
  for (int i=0; i<numElements; i++) printf("%d %f\n", i, a_host[i]);  

  // re-initialize host array to do the same on the gpu again
  for (int i=0; i<numElements; i++) a_host[i] = (float)i;
  printf("\nGPU-version:\n");

  square_array_gpu(a_host, numElements);

  // print results
  for (int i=0; i<numElements; i++) printf("%d %f\n", i, a_host[i]);
  
  // cleanup
  free(a_host);  

  printf("\nPress ENTER to exit...\n");
  getchar();
}
