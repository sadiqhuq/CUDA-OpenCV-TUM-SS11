#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    summer term 2011 / 19-26th September
*
* project: gradient
* file:    gradient.cu
*
* 
\********* PLEASE ENTER YOUR CORRECT STUDENT NAME AND ID BELOW **************/
const char* studentName = "Sadiq Huq";
const int   studentID   = 3273623;
/****************************************************************************\
*
* In this file the following methods have to be edited or completed:
*
* derivativeY_sm_d(const float *inputImage, ... )
* derivativeY_sm_d(const float3 *inputImage, ... )
* gradient_magnitude_d(const float *inputImage, ... )
* gradient_magnitude_d(const float3 *inputImage, ... )
*
\****************************************************************************/


#include <cutil.h>
#include <cutil_inline.h>
#include "gradient.cuh"



#define BW 16
#define BH 16



const char* getStudentName() { return studentName; };
int         getStudentID()   { return studentID; };
bool        checkStudentNameAndID() { return strcmp(studentName, "John Doe") != 0 && studentID != 1234567; }; 




__global__ void derivativeX_sm_d(const float *inputImage, float *outputImage,
                                 int iWidth, int iHeight, size_t iPitchBytes)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ float u[BW+2][BH];


  if (x < iWidth && y < iHeight) {
    u[threadIdx.x+1][threadIdx.y] = *((float*)((char*)inputImage + y*iPitchBytes)+x);

    if (x == 0) u[threadIdx.x][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
    else if (x == (iWidth-1)) u[threadIdx.x+2][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
    else {
      if (threadIdx.x == 0) u[threadIdx.x][threadIdx.y] = *((float*)((char*)inputImage + y*iPitchBytes)+x-1);
      else if (threadIdx.x == blockDim.x-1) u[threadIdx.x+2][threadIdx.y] = *((float*)((char*)inputImage + y*iPitchBytes)+x+1);
    }
  }

  __syncthreads();

  if (x < iWidth && y < iHeight)
    *((float*)(((char*)outputImage) + y*iPitchBytes)+ x) = 0.5f*(u[threadIdx.x+2][threadIdx.y]-u[threadIdx.x][threadIdx.y])+128;
}




__global__ void derivativeX_sm_d(const float3 *inputImage, float3 *outputImage,
                                 int iWidth, int iHeight, size_t iPitchBytes)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  float3 imgValue ;
  __shared__ float3 u[BW+2][BH];

  if (x < iWidth && y < iHeight) {
    u[threadIdx.x+1][threadIdx.y] = *((float3*)((char*)inputImage + y*iPitchBytes)+x);

    if (x == 0) u[threadIdx.x][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
    else if (x == (iWidth-1)) u[threadIdx.x+2][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
    else {
      if (threadIdx.x == 0) u[threadIdx.x][threadIdx.y] = *((float3*)((char*)inputImage + y*iPitchBytes)+x-1);
      else if (threadIdx.x == blockDim.x-1) u[threadIdx.x+2][threadIdx.y] = *((float3*)((char*)inputImage + y*iPitchBytes)+x+1);
    }
  }

  __syncthreads();

  // +128 to stay within range 255
  if (x < iWidth && y < iHeight) {
    imgValue.x = 0.5f*(u[threadIdx.x+2][threadIdx.y].x - u[threadIdx.x][threadIdx.y].x)+128;
    imgValue.y = 0.5f*(u[threadIdx.x+2][threadIdx.y].y - u[threadIdx.x][threadIdx.y].y)+128;
    imgValue.z = 0.5f*(u[threadIdx.x+2][threadIdx.y].z - u[threadIdx.x][threadIdx.y].z)+128;
    
    *((float3*)(((char*)outputImage) + y*iPitchBytes)+ x) = imgValue;
  }
  
}



__global__ void derivativeY_sm_d(const float *inputImage, float *outputImage,
                                 int iWidth, int iHeight, size_t iPitchBytes)
{

  // ### implement me ### 
	 const int x = blockIdx.x * blockDim.x + threadIdx.x;
	  const int y = blockIdx.y * blockDim.y + threadIdx.y;

	  __shared__ float u[BW][BH+2];


	  if (x < iWidth && y < iHeight) {
	    u[threadIdx.x][threadIdx.y+1] = *((float*)((char*)inputImage + y*iPitchBytes)+x);

	    if (y == 0) u[threadIdx.x][threadIdx.y] = u[threadIdx.x][threadIdx.y+1];
	    else if (y == (iHeight-1)) u[threadIdx.x][threadIdx.y+2] = u[threadIdx.x][threadIdx.y+1];
	    else {
	      if (threadIdx.y == 0) u[threadIdx.x][threadIdx.y] = *((float*)((char*)inputImage + (y-1)*iPitchBytes)+x);
	      else if (threadIdx.y == blockDim.y-1) u[threadIdx.x][threadIdx.y+2] = *((float*)((char*)inputImage + (y+1)*iPitchBytes)+x);
	    }
	  }

	  __syncthreads();

	  if (x < iWidth && y < iHeight)
	    *((float*)(((char*)outputImage) + y*iPitchBytes)+ x) = 0.5f*(u[threadIdx.x][threadIdx.y+2]-u[threadIdx.x][threadIdx.y])+128;

}



__global__ void derivativeY_sm_d(const float3 *inputImage, float3 *outputImage,
                                 int iWidth, int iHeight, size_t iPitchBytes)
{

 //  ### implement me ### 
	  const int x = blockIdx.x * blockDim.x + threadIdx.x;
	  const int y = blockIdx.y * blockDim.y + threadIdx.y;
	  float3 imgValue ;
	  __shared__ float3 u[BW][BH+2];

	  if (x < iWidth && y < iHeight) {
	    u[threadIdx.x][threadIdx.y+1] = *((float3*)((char*)inputImage + y*iPitchBytes)+x);

	    if (y == 0) u[threadIdx.x][threadIdx.y] = u[threadIdx.x][threadIdx.y+1];
	    else if (y == (iWidth-1)) u[threadIdx.x][threadIdx.y+2] = u[threadIdx.x][threadIdx.y+1];
	    else {
	      if (threadIdx.y == 0) u[threadIdx.x][threadIdx.y] = *((float3*)((char*)inputImage + (y-1)*iPitchBytes)+x);
	      else if (threadIdx.y == blockDim.y-1) u[threadIdx.x][threadIdx.y+2] = *((float3*)((char*)inputImage + (y+1)*iPitchBytes)+x);
	    }
	  }

	  __syncthreads();

	  // +128 to stay within range 255
	  if (x < iWidth && y < iHeight) {
	    imgValue.x = 0.5f*(u[threadIdx.x][threadIdx.y+2].x - u[threadIdx.x][threadIdx.y].x)+128;
	    imgValue.y = 0.5f*(u[threadIdx.x][threadIdx.y+2].y - u[threadIdx.x][threadIdx.y].y)+128;
	    imgValue.z = 0.5f*(u[threadIdx.x][threadIdx.y+2].z - u[threadIdx.x][threadIdx.y].z)+128;
//	    float3 value = make_float3(0.0f, 0.0f, 0.0f);
//	    *((float3*)(((char*)outputImage) + y*iPitchBytes)+ x) = value;
	    *((float3*)(((char*)outputImage) + y*iPitchBytes)+ x) = imgValue;
	  }
	  
}





__global__ void gradient_magnitude_d(const float *inputImage, float *outputImage,
                                     int iWidth, int iHeight, size_t iPitchBytes)
{

  // ### implement me ### 
	 const int x = blockIdx.x * blockDim.x + threadIdx.x;
	  const int y = blockIdx.y * blockDim.y + threadIdx.y;
	  
	  __shared__ float u[BW + 2][BH+2]; 

	  if (x < iWidth && y < iHeight) 
	  {
	    u[threadIdx.x + 1][threadIdx.y] = *((float*)((char*)inputImage + y*iPitchBytes) + x);
	    u[threadIdx.x][threadIdx.y+1] = *((float*)((char*)inputImage + y*iPitchBytes) + x);

	    // BC for X
	    if (x == 0) // clamp left border 
	    	u[threadIdx.x][threadIdx.y+1] = u[threadIdx.x+1][threadIdx.y];
	    else if (x == (iWidth-1)) // clamp right
	    	u[threadIdx.x+2][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
	    else // interier pixels
	    {
	      if (threadIdx.x == 0) 
	      	u[threadIdx.x][threadIdx.y] = *((float*)((char*)inputImage + y*iPitchBytes)+x-1);
	      else if (threadIdx.x == blockDim.x-1) 
	      	u[threadIdx.x+2][threadIdx.y] = *((float*)((char*)inputImage + y*iPitchBytes)+x+1);
	    }
	    
	    // BC for Y
	    if (y == 0) // clamp left border 
	    	u[threadIdx.x + 1][threadIdx.y] = u[threadIdx.x][threadIdx.y+1];
	    else if (y == (iHeight-1)) // clamp right
	    	u[threadIdx.x + 1][threadIdx.y+2] = u[threadIdx.x + 1][threadIdx.y+1];
	    else // interior 
	    {
	      if (threadIdx.y == 0) 
	      	u[threadIdx.x ][threadIdx.y] = *((float*)((char*)inputImage + (y-1)*iPitchBytes)+x);
	      else if (threadIdx.y == blockDim.y-1) 
	      	u[threadIdx.x ][threadIdx.y+2] = *((float*)((char*)inputImage + (y+1)*iPitchBytes)+x);
	    }        
	  }  
	 
	  __syncthreads();  

	  if (x < iWidth && y < iHeight)
	  {
		  
		  float dx =  0.5f*(u[threadIdx.x+2][threadIdx.y]-u[threadIdx.x][threadIdx.y])+128;
		  float dy = 0.5f*(u[threadIdx.x][threadIdx.y+2]-u[threadIdx.x][threadIdx.y])+128;
	   
	    *((float*)(((char*)outputImage) + y * iPitchBytes) + x) =  	sqrt(dx * dx + dy * dy);
	  }
}





__global__ void gradient_magnitude_d(const float3 *inputImage, float3 *outputImage,
                                     int iWidth, int iHeight, size_t iPitchBytes)
{

  // ### implement me ### 

	  const int x = blockIdx.x * blockDim.x + threadIdx.x;
	  const int y = blockIdx.y * blockDim.y + threadIdx.y;
	  float3 imgValue ;
	  __shared__ float3 u[BW+2][BH+2];
	
	
	  if (x < iWidth && y < iHeight) {
		  
	    u[threadIdx.x+1][threadIdx.y] = *((float3*)((char*)inputImage + y*iPitchBytes)+x);
	    u[threadIdx.x][threadIdx.y+1] = *((float3*)((char*)inputImage + y*iPitchBytes)+x);

	    // BC for X
	    if (x == 0) u[threadIdx.x][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
	    else if (x == (iWidth-1)) u[threadIdx.x+2][threadIdx.y] = u[threadIdx.x+1][threadIdx.y];
	    else {
	      if (threadIdx.x == 0) u[threadIdx.x][threadIdx.y] = *((float3*)((char*)inputImage + y*iPitchBytes)+x-1);
	      else if (threadIdx.x == blockDim.x-1) u[threadIdx.x+2][threadIdx.y] = *((float3*)((char*)inputImage + y*iPitchBytes)+x+1);
	    }
	    
	   // BC for Y
		  if (y == 0) u[threadIdx.x][threadIdx.y] = u[threadIdx.x][threadIdx.y+1];
		 	    else if (y == (iHeight-1)) u[threadIdx.x][threadIdx.y+2] = u[threadIdx.x][threadIdx.y+1];
		 	    else {
		 	      if (threadIdx.y == 0) u[threadIdx.x][threadIdx.y] = *((float3*)((char*)inputImage + (y-1)*iPitchBytes)+x);
		 	      else if (threadIdx.y == blockDim.y-1) u[threadIdx.x][threadIdx.y+2] = *((float3*)((char*)inputImage + (y+1)*iPitchBytes)+x);
		 	    }
	  }

	  __syncthreads();

	
	  if (x < iWidth && y < iHeight) 
	  {
		  float3 dx3 = make_float3(
				  0.5f*(u[threadIdx.x+2][threadIdx.y].x - u[threadIdx.x][threadIdx.y].x)+128,
				  0.5f*(u[threadIdx.x+2][threadIdx.y].y - u[threadIdx.x][threadIdx.y].y)+128,
				  0.5f*(u[threadIdx.x+2][threadIdx.y].z - u[threadIdx.x][threadIdx.y].z)+128);

		  float3 dy3 = make_float3(
				  0.5f*(u[threadIdx.x][threadIdx.y+2].x - u[threadIdx.x][threadIdx.y].x)+128,
				  0.5f*(u[threadIdx.x][threadIdx.y+2].y - u[threadIdx.x][threadIdx.y].y)+128,
				  0.5f*(u[threadIdx.x][threadIdx.y+2].z - u[threadIdx.x][threadIdx.y].z)+128 );

		  float3 value = make_float3( 
				  sqrt(dx3.x * dx3.x + dy3.x * dy3.x ),
				  sqrt(dx3.y * dx3.z + dy3.y * dy3.y ),
				  sqrt(dx3.z * dx3.z + dy3.z * dy3.z ));
		  
		  *((float3*)(((char*)outputImage) + y*iPitchBytes)+ x) = value;
  
		  //	 	    *((float3*)(((char*)outputImage) + y*iPitchBytes)+ x) = sqrt(dx3*dx3 + dy3*dy3);
	  }
	
	
	
	
	
}



void gpu_derivative_sm_d(const float *inputImage, float *outputImage,
                         int iWidth, int iHeight, int iSpectrum, int mode)
{
  size_t iPitchBytes;
  float *inputImage_d = 0, *outputImage_d = 0;

  dim3 blockSize(BW, BH);  
  dim3 gridSize( (int)ceil(iWidth/(float)BW), (int)ceil(iHeight/(float)BH) );
  //dim3 smSize(BW+2,BH);

  if(iSpectrum == 1) {
    cutilSafeCall( hipMallocPitch( (void**)&(inputImage_d), &iPitchBytes, iWidth*sizeof(float), iHeight ) );
    cutilSafeCall( hipMallocPitch( (void**)&(outputImage_d), &iPitchBytes, iWidth*sizeof(float), iHeight ) );

    cutilSafeCall( hipMemcpy2D(inputImage_d, iPitchBytes, inputImage, iWidth*sizeof(float), iWidth*sizeof(float), iHeight, hipMemcpyHostToDevice) );

    if (mode == 0)
      derivativeX_sm_d<<<gridSize, blockSize>>>(inputImage_d, outputImage_d, iWidth, iHeight, iPitchBytes);
    else if (mode == 1)
      derivativeY_sm_d<<<gridSize, blockSize>>>(inputImage_d, outputImage_d, iWidth, iHeight, iPitchBytes);
    else if (mode == 2)
      gradient_magnitude_d<<<gridSize, blockSize>>>(inputImage_d, outputImage_d, iWidth, iHeight, iPitchBytes);

    cutilSafeCall( hipDeviceSynchronize() );
    cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float), outputImage_d, iPitchBytes, iWidth*sizeof(float), iHeight, hipMemcpyDeviceToHost) );
  }
  else if(iSpectrum == 3) {
    cutilSafeCall( hipMallocPitch( (void**)&(inputImage_d), &iPitchBytes, iWidth*sizeof(float3), iHeight ) );
    cutilSafeCall( hipMallocPitch( (void**)&(outputImage_d), &iPitchBytes, iWidth*sizeof(float3), iHeight ) );

    cutilSafeCall( hipMemcpy2D(inputImage_d, iPitchBytes, inputImage, iWidth*sizeof(float3), iWidth*sizeof(float3), iHeight, hipMemcpyHostToDevice) );

    if (mode == 0)
      derivativeX_sm_d<<<gridSize, blockSize>>>((float3*)inputImage_d, (float3*)outputImage_d, iWidth, iHeight, iPitchBytes);
    else if (mode == 1)
      derivativeY_sm_d<<<gridSize, blockSize>>>((float3*)inputImage_d, (float3*)outputImage_d, iWidth, iHeight, iPitchBytes);
    else if (mode == 2)
      gradient_magnitude_d<<<gridSize, blockSize>>>((float3*)inputImage_d, (float3*)outputImage_d, iWidth, iHeight, iPitchBytes);

    cutilSafeCall( hipDeviceSynchronize() );
    cutilSafeCall( hipMemcpy2D(outputImage, iWidth*sizeof(float3), outputImage_d, iPitchBytes, iWidth*sizeof(float3), iHeight, hipMemcpyDeviceToHost) );
  }

  cutilSafeCall( hipFree(inputImage_d) );
  cutilSafeCall( hipFree(outputImage_d) );
}
